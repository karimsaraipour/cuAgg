#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdlib>
#include <math.h>
#include <utility>

#include "../src/cuda.cuh"
#include "../src/graph/generator.h"
#include "../src/graph/graph.h"
#include "../src/kernels/aggregate.cuh"

bool feq(float f1, float f2) { return fabs(f1 - f2) < 0.001; }

void aggregate_cpu_oracle(const GraphPtr g, const FeatureVec &in_features,
                          FeatureVec &out_features, int num_features) {
  FeatureVec node_features(num_features);

  for (NodeT v = 0; v < g->num_nodes; v++) {
    // Reset node features
    for (IndexT f = 0; f < num_features; f++)
      node_features[f] = in_features[v * num_features + f];

    // Aggregate features
    for (IndexT i = g->index[v]; i < g->index[v + 1]; i++) {
      NodeT u = g->neighbors[i];
      for (IndexT f = 0; f < num_features; f++)
        node_features[f] += in_features[u * num_features + f];
    }

    // Write out
    for (IndexT f = 0; f < num_features; f++)
      out_features[v * num_features + f] = node_features[f];
  }
}

int main() {
  constexpr int TEST_SCALE = 14;
  constexpr int TEST_DEGREE = 10;
  constexpr IndexT TEST_NUM_FEATURES = 1024;

  constexpr int BLOCK_DIM_X = 16;
  constexpr int BLOCK_DIM_Y = 32;

  // Generate graph
  auto g = generate_krongraph(TEST_SCALE, TEST_DEGREE);
  assert(g != nullptr && "graph is invalid");

  // Get CPU oracle (single-threaded)
  auto features = generate_features(g->num_nodes, TEST_NUM_FEATURES);
  assert(!features.empty() && "features are empty");
  FeatureVec oracle_features(features.size());

  aggregate_cpu_oracle(g, features, oracle_features, TEST_NUM_FEATURES);

  // Get GPU aggregated features
  IndexT *cu_index;
  NodeT *cu_neighbors;
  FeatureT *cu_in_features;
  FeatureT *cu_out_features;
  size_t size_index = g->index.size() * sizeof(IndexT);
  size_t size_neighbors = g->neighbors.size() * sizeof(NodeT);
  size_t size_features = features.size() * sizeof(FeatureT);
  CUDA_ERRCHK(hipMalloc((void **)&cu_index, size_index));
  CUDA_ERRCHK(hipMalloc((void **)&cu_neighbors, size_neighbors));
  CUDA_ERRCHK(hipMalloc((void **)&cu_in_features, size_features));
  CUDA_ERRCHK(hipMalloc((void **)&cu_out_features, size_features));
  CUDA_ERRCHK(hipMemcpy(cu_index, g->index.data(), size_index,
                         hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cu_neighbors, g->neighbors.data(), size_neighbors,
                         hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemcpy(cu_in_features, features.data(), size_features,
                         hipMemcpyHostToDevice));
  CUDA_ERRCHK(hipMemset(cu_out_features, 0, size_features));

  dim3 dim_block(BLOCK_DIM_X, BLOCK_DIM_Y);
  dim3 dim_grid((g->num_nodes + BLOCK_DIM_X - 1) / BLOCK_DIM_X,
                (TEST_NUM_FEATURES + BLOCK_DIM_Y - 1) / BLOCK_DIM_Y);

  aggregate_naive<<<dim_grid, dim_block>>>(cu_index, cu_neighbors,
                                           cu_in_features, cu_out_features,
                                           g->num_nodes, TEST_NUM_FEATURES);

  // Copy results to CPU memory
  FeatureT *test_features = new FeatureT[features.size()];
  CUDA_ERRCHK(hipMemcpy(test_features, cu_out_features, size_features,
                         hipMemcpyDeviceToHost));

  for (size_t i = 0; i < features.size(); i++)
    assert(feq(test_features[i], oracle_features[i]) && "features don't match");

  delete[] test_features;

  return EXIT_SUCCESS;
}
