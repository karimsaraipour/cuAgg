#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdlib>
#include <math.h>
#include <utility>

#include "../src/graph/generator.h"
#include "../src/graph/graph.h"
#include "../src/kernels/aggregate.cuh"

bool feq(float f1, float f2) { return fabs(f1 - f2) < 0.001; }

void aggregate_cpu_oracle(const GraphPtr g, const FeatureVec &in_features,
                          FeatureVec &out_features, int num_features) {
  FeatureVec node_features(num_features);

  for (NodeT v = 0; v < g->num_nodes; v++) {
    // Reset node features
    for (IndexT f = 0; f < num_features; f++)
      node_features[f] = in_features[v * num_features + f];

    // Aggregate features
    for (IndexT i = g->index[v]; i < g->index[v + 1]; i++) {
      NodeT u = g->neighbors[i];
      for (int f = 0; f < num_features; f++)
        node_features[f] += in_features[u * num_features + f];
    }

    // Write out
    for (IndexT f = 0; f < num_features; f++)
      out_features[v * num_features + f] = node_features[f];
  }
}

int main() {
  constexpr int TEST_SCALE = 14;
  constexpr int TEST_DEGREE = 10;
  constexpr IndexT TEST_NUM_FEATURES = 1024;

  // Generate graph
  auto g = generate_krongraph(TEST_SCALE, TEST_DEGREE);
  assert(g != nullptr && "graph is invalid");

  // Get CPU oracle (single-threaded)
  auto features = generate_features(g->num_nodes, TEST_NUM_FEATURES);
  assert(!features.empty() && "features are empty");
  FeatureVec oracle_features(features.size());
  FeatureVec test_features(features.size());

  aggregate_cpu_oracle(g, features, oracle_features, TEST_NUM_FEATURES);

  // Get CPU parallel aggregated features
  aggregate_cpu(g, features, test_features, TEST_NUM_FEATURES);

  for (size_t i = 0; i < features.size(); i++)
    assert(feq(test_features[i], oracle_features[i]) && "features don't match");

  return EXIT_SUCCESS;
}
