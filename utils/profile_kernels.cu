#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>

#include "../src/graph/graph.h"
#include "../src/graph/partition.h"
#include "../src/kernels/aggregate.cuh"

int main(int argc, char *argv[]) {
  if (argc < 3 && argc > 5) {
    std::cerr << "Usage: " << argv[0]
              << " <graph_file> <num_features> [<num_buffers> [<sparsity>]]"
              << std::endl;
    return EXIT_FAILURE;
  }

  // Load graph
  auto g = GraphPtr(new Graph());
  std::ifstream ifs(argv[1], std::ifstream::in);
  ifs >> *g;
  ifs.close();

  // Compute tile size
  IndexT num_features = atoi(argv[2]);
  int db_size = (argc >= 4) ? atoi(argv[3]) : 2;
  float sparsity = (argc >= 5) ? atof(argv[4]) : 0.2f;

  NodeT tile_size = std::min(
      get_square_tile_size(num_features, db_size, sparsity), g->num_idx_nodes);

  // Partition graph and report how many actually meet sparsity requirement
  auto partitions = partition_square_tile(g, tile_size);

  size_t w_tile_size = tile_size;
  size_t valid_ngh_size = w_tile_size * w_tile_size * sparsity;
  std::cout << valid_ngh_size << std::endl;
  NodeT failed_tiles = 0;
  for (auto &part : partitions) {
    IndexT num_edges = part.subgraph->index[part.subgraph->num_idx_nodes];
    float tile_sparsity =
        static_cast<float>(num_edges) / (w_tile_size * w_tile_size);
    std::cout << num_edges << ' ' << tile_sparsity << std::endl;
    if (part.subgraph->neighbors.size() > valid_ngh_size)
      failed_tiles++;
  }

  auto num_nodes = g->num_idx_nodes;
  auto num_edges = g->index[g->num_idx_nodes];
  std::cout << "Graph" << std::endl
            << "  # of nodes: " << num_nodes << std::endl
            << "  # of edges: " << num_edges << std::endl
            << "  avg degree: " << (double)num_edges / num_nodes << std::endl;
  std::cout << std::endl;
  std::cout << "Tile" << std::endl
            << "  Tile size: " << tile_size << std::endl
            << "  Total tiles: " << partitions.size() << std::endl
            << "  Failed tiles: " << failed_tiles << std::endl;

  return EXIT_SUCCESS;
}
