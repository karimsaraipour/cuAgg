#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <sstream>

#include "../src/cuda.cuh"
#include "../src/graph/generator.h"
#include "../src/graph/graph.h"
#include "../src/graph/partition.h"
#include "../src/kernels/aggregate.cuh"
#include "../src/kernels/aggregate_templated.cuh"

int main(int argc, char *argv[]) {
  if (argc < 3 && argc > 5) {
    std::cerr << "Usage: " << argv[0]
              << " <graph_file> <num_features> [<num_buffers> [<sparsity>]]"
              << std::endl;
    return EXIT_FAILURE;
  }

  // Load graph
  auto g = GraphPtr(new Graph());
  std::ifstream ifs(argv[1], std::ifstream::in);
  ifs >> *g;
  ifs.close();

  // Compute tile size
  IndexT num_features = atoi(argv[2]);
  int db_size = (argc >= 4) ? atoi(argv[3]) : 2;
  float sparsity = (argc >= 5) ? atof(argv[4]) : 0.2f;

  NodeT tile_size = std::min(
      get_square_tile_size(num_features, db_size, sparsity), g->num_idx_nodes);

  // Partition graph and report how many actually meet sparsity requirement
  auto partitions = partition_square_tile(g, tile_size);

  size_t w_tile_size = tile_size;
  size_t valid_ngh_size = w_tile_size * w_tile_size * sparsity;
  NodeT failed_tiles = 0;
  for (auto &part : partitions) {
    IndexT num_edges = part.subgraph->index[part.subgraph->num_idx_nodes];
    float tile_sparsity =
        static_cast<float>(num_edges) / (w_tile_size * w_tile_size);
    std::cout << num_edges << ' ' << tile_sparsity << std::endl;
    if (part.subgraph->neighbors.size() > valid_ngh_size)
      failed_tiles++;
  }

  // Print stats
  auto num_nodes = g->num_idx_nodes;
  auto num_edges = g->index[g->num_idx_nodes];
  std::cout << "Graph" << std::endl
            << "  # of nodes: " << num_nodes << std::endl
            << "  # of edges: " << num_edges << std::endl
            << "  avg degree: " << (double)num_edges / num_nodes << std::endl;
  std::cout << std::endl;
  std::cout << "Tile" << std::endl
            << "  Tile size: " << tile_size << std::endl
            << "  Total tiles: " << partitions.size() << std::endl
            << "  Failed tiles: " << failed_tiles << std::endl;

  // Generate profile
  auto features = generate_features(g->num_idx_nodes, num_features);
  assert(!features.empty() && "features are empty");
  std::cout << "Features created" << std::endl;

  FeatureT *dummy_features = new FeatureT[features.size()];
  NodeT num_tiles1D = (g->num_idx_nodes + tile_size - 1) / tile_size;

  // Generate timing events
  hipEvent_t *starts = new hipEvent_t[partitions.size()];
  hipEvent_t *stops = new hipEvent_t[partitions.size()];
  for (size_t i = 0; i < partitions.size(); i++) {
    CUDA_ERRCHK(hipEventCreate(&starts[i]));
    CUDA_ERRCHK(hipEventCreate(&stops[i]));
  }
  IndexT e = 0;

  // Run kernel
  aggregate_double_buffer_naive(
      partitions, num_tiles1D, features, dummy_features, num_features,
      tile_size,
      [&starts, &stops,
       &e](const IndexT *const index, const NodeT *const neighbors,
           const FeatureT *const in_features, FeatureT *const out_features,
           const NodeT num_nodes, const IndexT num_features) -> void {
        hipEventRecord(starts[e]);
        aggregate_dyn<<<num_nodes, 32>>>(index, neighbors, in_features,
                                         out_features, num_nodes, num_features);
        hipEventRecord(stops[e]);
        e++;
      },
      db_size, valid_ngh_size);

  // Report times
  for (size_t i = 0; i < partitions.size(); i++) {
    CUDA_ERRCHK(hipEventSynchronize(stops[i]));
    float elapsed;
    CUDA_ERRCHK(hipEventElapsedTime(&elapsed, starts[i], stops[i]));
    std::cout << "  Time: " << elapsed << " ms" << std::endl;
  }

  delete[] dummy_features;

  return EXIT_SUCCESS;
}
